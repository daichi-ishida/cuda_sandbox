#include "kernel.h"

#include <hip/hip_runtime.h>

#include <iostream>


void cuda_main()
{
    hipError_t cudaStatus = hipSetDevice(0);

    size_t free, total, use;

    // Before memory allocation
    hipMemGetInfo(&free, &total);
    use = total - free;
    use /= 1024 * 1024;
    std::cout << "[before allocation] use:  " << use << " [MB]\n";

    int N = 52428800; // 200MB
    float *x;
    size_t memsize = sizeof(float) * N;

    hipMalloc(&x, memsize);

    // After memory allocation.
    hipMemGetInfo(&free, &total);
    use = total - free;
    use /= 1024 * 1024;
    std::cout << "[after allocation] use:   " << use << " [MB]\n";

    hipFree(x);

    // After release memory.
    hipMemGetInfo(&free, &total);
    use = total - free;
    use /= 1024 * 1024;
    std::cout << "[after release] use:      " << use << " [MB]\n";
}